#include "hip/hip_runtime.h"
#include "buffer.h"

void Buffer::init(Buffer& buf, int num, int dimention) {
    buf.num = num;
    buf.dimention = dimention;
    CUDA_ERROR_CHECK(hipMalloc(&buf.buffer, buf.Size()));
}

void Buffer::init(Buffer& buf, Buffer& src, int dimention) {
    Buffer::init(buf, src.num, dimention);
}

void Buffer::copy(Buffer& dst, Buffer& src) {
    CUDA_ERROR_CHECK(hipMemcpy(dst.buffer, src.buffer, dst.Size(), hipMemcpyDeviceToDevice));
}

void Buffer::copy(Buffer& dst, float* src) {
    CUDA_ERROR_CHECK(hipMemcpy(dst.buffer, src, dst.Size(), hipMemcpyHostToDevice));
}

void Buffer::copy(float* dst, Buffer& src) {
    CUDA_ERROR_CHECK(hipMemcpy(dst, src.buffer, src.Size(), hipMemcpyDeviceToHost));
}

__global__ void BufferLinerKernel(float* buffer, float w, float b, int width, int height) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height)return;
    int pidx = px + width * py;
    buffer[pidx] = buffer[pidx] * w + b;
}

void Buffer::liner(Buffer& buf, float w, float b) {
    dim3 block = getBlock(buf.num, buf.dimention);
    dim3 grid = getGrid(block, buf.num, buf.dimention);
    void* args[] = { &buf.buffer, &w,&b,&buf.num,&buf.dimention };
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(BufferLinerKernel), grid, block, args, 0, NULL));
}

__global__ void BufferRandomKernel(float* buffer, float min, float max, int width, int height, unsigned int seed) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height)return;
    int pidx = px + width * py;
    buffer[pidx] += min + (max - min) * getUniform(pidx, seed, 0xba5ec0de);
}

void Buffer::addRandom(Buffer& buf, float min, float max) {
    unsigned int seed = rand();
    dim3 block = getBlock(buf.num, buf.dimention);
    dim3 grid = getGrid(block, buf.num, buf.dimention);
    void* args[] = { &buf.buffer,&min,&max,&buf.num,&buf.dimention, &seed };
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(BufferRandomKernel), grid, block, args, 0, NULL));
}

__global__ void BufferClampKernel(float* buffer, float min, float max, int width, int height) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height)return;
    int pidx = px + width * py;
    buffer[pidx] = clamp(buffer[pidx],min, max);
}

__global__ void BufferStepKernel(float* buffer, float threshold, int width, int height) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height)return;
    int pidx = px + width * py;
    buffer[pidx] = buffer[pidx] < threshold ? 0.f : 1.f;
}

void Buffer::clamp(Buffer& buf, float min, float max) {
    unsigned int seed = rand();
    dim3 block = getBlock(buf.num, buf.dimention);
    dim3 grid = getGrid(block, buf.num, buf.dimention);
    void* args[] = { &buf.buffer,&min,&max,&buf.num,&buf.dimention };
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(BufferClampKernel), grid, block, args, 0, NULL));
}


void BufferGrad::init(BufferGrad& buf, int num, int dimention) {
    Buffer::init(buf, num, dimention);
    CUDA_ERROR_CHECK(hipMalloc(&buf.grad, buf.Size()));
}

void BufferGrad::init(BufferGrad& buf, Buffer& src, int dimention) {
    init(buf, src.num, dimention);
}

void BufferGrad::clear(BufferGrad& buf) {
    CUDA_ERROR_CHECK(hipMemset(buf.grad, 0, buf.Size()));
}



void Attribute::init(Attribute& attr, int vboNum, int vaoNum, int dimention) {
    attr.dimention = dimention;
    attr.vboNum = vboNum;
    attr.vaoNum = vaoNum;
    CUDA_ERROR_CHECK(hipMalloc(&attr.vbo, attr.vboSize()));
    CUDA_ERROR_CHECK(hipMalloc(&attr.vao, attr.vaoSize()));
}

void Attribute::init(Attribute& attr, Attribute& src, int dimention) {
    attr.dimention = dimention;
    attr.vboNum = src.vboNum;
    CUDA_ERROR_CHECK(hipMalloc(&attr.vbo, attr.vboSize()));
    attr.vaoNum = src.vaoNum;
    attr.vao = src.vao;
}

void Attribute::loadOBJ(const char* path, Attribute* pos, Attribute* texel, Attribute* normal) {
    FILE* file = fopen(path, "r");
    if (file == NULL) {
        printf("Impossible to open the file !\n");
        return;
    }

    std::vector<float> tempPos, tempTexel, tempNormal;
    std::vector<unsigned int> tempPosIndex, tempTexelIndex, tempNormalIndex;
    int posNum = 0, texelNum = 0, normalNum = 0, indexNum = 0;
    while (1) {
        char lineHeader[128];
        int res = fscanf(file, "%s", lineHeader);
        if (res == EOF)
            break;
        if (strcmp(lineHeader, "v") == 0) {
            float v[3];
            fscanf(file, "%f %f %f\n", &v[0], &v[1], &v[2]);
            tempPos.push_back(v[0]);
            tempPos.push_back(v[1]);
            tempPos.push_back(v[2]);
            posNum++;
        }
        else if (strcmp(lineHeader, "vt") == 0) {
            float v[2];
            fscanf(file, "%f %f\n", &v[0], &v[1]);
            tempTexel.push_back(v[0]);
            tempTexel.push_back(v[1]);
            texelNum++;
        }
        else if (strcmp(lineHeader, "vn") == 0) {
            float v[3];
            fscanf(file, "%f %f %f\n", &v[0], &v[1], &v[2]);
            tempNormal.push_back(v[0]);
            tempNormal.push_back(v[1]);
            tempNormal.push_back(v[2]);
            normalNum++;
        }
        else if (strcmp(lineHeader, "f") == 0 && posNum > 0) {
            unsigned int idx[9];
            if (texelNum > 0 && normalNum > 0) {
                int matches = fscanf(file, "%d/%d/%d %d/%d/%d %d/%d/%d\n", &idx[0], &idx[3], &idx[6], &idx[1], &idx[4], &idx[7], &idx[2], &idx[5], &idx[8]);
                if (matches != 9) {
                    printf("File can't be read by our simple parser : ( Try exporting with other options\n");
                    return;
                }
                tempTexelIndex.push_back(idx[3] - 1);
                tempTexelIndex.push_back(idx[4] - 1);
                tempTexelIndex.push_back(idx[5] - 1);
                tempNormalIndex.push_back(idx[6] - 1);
                tempNormalIndex.push_back(idx[7] - 1);
                tempNormalIndex.push_back(idx[8] - 1);
            }
            else if (texelNum > 0) {
                int matches = fscanf(file, "%d/%d %d/%d %d/%d\n", &idx[0], &idx[3], &idx[1], &idx[4], &idx[2], &idx[5]);
                if (matches != 6) {
                    printf("File can't be read by our simple parser : ( Try exporting with other options\n");
                    return;
                }
                tempTexelIndex.push_back(idx[3] - 1);
                tempTexelIndex.push_back(idx[4] - 1);
                tempTexelIndex.push_back(idx[5] - 1);
            }
            else if (normalNum > 0) {
                int matches = fscanf(file, "%d//%d %d//%d %d//%d\n", &idx[0], &idx[6], &idx[1], &idx[7], &idx[2], &idx[8]);
                if (matches != 6) {
                    printf("File can't be read by our simple parser : ( Try exporting with other options\n");
                    return;
                }
                tempNormalIndex.push_back(idx[6] - 1);
                tempNormalIndex.push_back(idx[7] - 1);
                tempNormalIndex.push_back(idx[8] - 1);
            }
            else {
                int matches = fscanf(file, "%d %d %d\n", &idx[0], &idx[1], &idx[2]);
                if (matches != 3) {
                    printf("File can't be read by our simple parser : ( Try exporting with other options\n");
                    return;
                }
            }
            tempPosIndex.push_back(idx[0] - 1);
            tempPosIndex.push_back(idx[1] - 1);
            tempPosIndex.push_back(idx[2] - 1);
            indexNum++;
        }
    }


    if (posNum > 0 && pos != nullptr) {
        Attribute::init(*pos, posNum, indexNum, 3);
        CUDA_ERROR_CHECK(hipMemcpy(pos->vbo, tempPos.data(), pos->vboSize(), hipMemcpyHostToDevice));
        CUDA_ERROR_CHECK(hipMemcpy(pos->vao, tempPosIndex.data(), pos->vaoSize(), hipMemcpyHostToDevice));
    }
    if (texelNum > 0 && texel != nullptr) {
        Attribute::init(*texel, texelNum, indexNum, 2);
        CUDA_ERROR_CHECK(hipMemcpy(texel->vbo, tempTexel.data(), texel->vboSize(), hipMemcpyHostToDevice));
        CUDA_ERROR_CHECK(hipMemcpy(texel->vao, tempTexelIndex.data(), texel->vaoSize(), hipMemcpyHostToDevice));
    }
    if (normalNum > 0 && normal != nullptr) {
        Attribute::init(*normal, normalNum, indexNum, 3);
        CUDA_ERROR_CHECK(hipMemcpy(normal->vbo, tempNormal.data(), normal->vboSize(), hipMemcpyHostToDevice));;
        CUDA_ERROR_CHECK(hipMemcpy(normal->vao, tempNormalIndex.data(), normal->vaoSize(), hipMemcpyHostToDevice));
    }
}

void Attribute::copy(Attribute& dst, Attribute& src) {
    hipMemcpy(dst.vbo, src.vbo, dst.vboSize(), hipMemcpyDeviceToDevice);
    if(dst.vao!=src.vao)hipMemcpy(dst.vao, src.vao, dst.vaoSize(), hipMemcpyDeviceToDevice);
}

void Attribute::liner(Attribute& attr, float w, float b) {
    dim3 block = getBlock(attr.vboNum, attr.dimention);
    dim3 grid = getGrid(block, attr.vboNum, attr.dimention);
    void* args[] = { &attr.vbo, &w,&b,&attr.vboNum,&attr.dimention };
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(BufferLinerKernel), grid, block, args, 0, NULL));
}

__global__ void distanceErrorKernel(const Attribute predict, const Attribute target, float* sum) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    if (px >= predict.vboNum)return;
    float d = 0.f;
    for (int i = 0; i < predict.dimention; i++) {
        float diff = predict.vbo[px * predict.dimention + i] - target.vbo[px * target.dimention + i];
        d += diff * diff;
    }
    atomicAdd(sum, sqrt(d));
}

float Attribute::distanceError(Attribute& predict, Attribute& target) {
    dim3 block = getBlock(predict.vboNum, 1);
    dim3 grid = getGrid(block, predict.vboNum, 1);
    float* dev;
    CUDA_ERROR_CHECK(hipMalloc(&dev, sizeof(float)));
    CUDA_ERROR_CHECK(hipMemset(dev, 0, sizeof(float)));
    void* args[] = { &predict, &target, &dev};
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(distanceErrorKernel), grid, block, args, 0, NULL));
    float sum;
    CUDA_ERROR_CHECK(hipMemcpy(&sum, dev, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_ERROR_CHECK(hipFree(dev));
    return sum / float(predict.vboNum);
}

void Attribute::addRandom(Attribute& attr, float min, float max) {
    unsigned int seed = rand();
    dim3 block = getBlock(attr.vboNum, attr.dimention);
    dim3 grid = getGrid(block, attr.vboNum, attr.dimention);
    void* args[] = { &attr.vbo,&min,&max,&attr.vboNum,&attr.dimention, &seed };
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(BufferRandomKernel), grid, block, args, 0, NULL));
}

void Attribute::step(Attribute& attr, float threshold) {
    dim3 block = getBlock(attr.vboNum, attr.dimention);
    dim3 grid = getGrid(block, attr.vboNum, attr.dimention);
    void* args[] = { &attr.vbo, &threshold,&attr.vboNum,&attr.dimention };
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(BufferStepKernel), grid, block, args, 0, NULL));
}

void AttributeGrad::init(AttributeGrad& attr, int vboNum, int vaoNum, int dimention) {
    Attribute::init(attr, vboNum, vaoNum, dimention);
    CUDA_ERROR_CHECK(hipMalloc(&attr.grad, attr.vboSize()));
}

void AttributeGrad::init(AttributeGrad& attr, Attribute& src, int dimention) {
    Attribute::init(attr, src, dimention);
    CUDA_ERROR_CHECK(hipMalloc(&attr.grad, attr.vboSize()));
}

void AttributeGrad::clear(AttributeGrad& attr) {
    CUDA_ERROR_CHECK(hipMemset(attr.grad, 0, attr.vboSize()));
}



void Texture::init(Texture& texture, int width, int height, int channel, int miplevel){
    int maxlevel = LSB(width | height) + 1;
    if (maxlevel > TEX_MAX_MIP_LEVEL)maxlevel = TEX_MAX_MIP_LEVEL;
    texture.width = width;
    texture.height = height;
    texture.channel = channel;
    texture.miplevel = miplevel < 1 ? 1 : (maxlevel < miplevel ? maxlevel : miplevel);
    int w = width, h = height;
    for (int i = 0; i < texture.miplevel; i++) {
        CUDA_ERROR_CHECK(hipMalloc(&texture.texture[i], (size_t)w * h * channel * sizeof(float)));
        w >>= 1; h >>= 1;
    }
};

void Texture::init(Texture& texture, float* original, int width, int height, int channel, int miplevel){
    int maxlevel = LSB(width | height) + 1;
    if (maxlevel > TEX_MAX_MIP_LEVEL)maxlevel = TEX_MAX_MIP_LEVEL;
    texture.width = width;
    texture.height = height;
    texture.channel = channel;
    texture.miplevel = miplevel < 1 ? 1 : (maxlevel < miplevel ? maxlevel : miplevel);
    texture.texture[0] = original;
    int w = width, h = height;
    for (int i = 1; i < texture.miplevel; i++) {
        CUDA_ERROR_CHECK(hipMalloc(&texture.texture[i], (size_t)w * h * channel * sizeof(float)));
        w >>= 1; h >>= 1;
    }
    buildMIP(texture);
};


__global__ void bilinearDownsamplingkernel(const Texture texture, int index, int width, int height) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= width || py >= height)return;
    int pidx = px + width * (py + height * pz);
    px <<= 1; py <<= 1;
    width <<= 1; height <<= 1;
    int xs = px > 0 ? -1 : 0;
    int xe = px < width - 2 ? 2 : 1;
    int ys = py > 0 ? -1 : 0;
    int ye = py < height - 2 ? 2 : 1;

    int idx = index - 1;
    float filter[4] = { .125f,.375f,.375f,.125f };
    for (int i = 0; i < 3; i++)texture.texture[index][pidx * texture.channel + i] = 0.f;
    for (int x = xs; x <= xe; x++) {
        for (int y = ys; y <= ye; y++) {
            float f = filter[x + 1] * filter[y + 1];
            int p = (px + x) + width * (py + y);
            for (int i = 0; i < 3; i++) {
                texture.texture[index][pidx * texture.channel + i] += texture.texture[idx][p * texture.channel + i] * f;
            }
        }
    }
}

void Texture::bilinearDownsampling(Texture& texture) {
    int i = 1;
    int w = texture.width, h = texture.height;
    void* args[] = { &texture, &i, &w, &h };
    for (; i < texture.miplevel; i++) {
        w >>= 1; h >>= 1;
        dim3 block = getBlock(w, h);
        dim3 grid = getGrid(block, w, h);
        CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(bilinearDownsamplingkernel), grid, block, args, 0, NULL));
    }
}

__global__ void downSampling(const Texture texture, int index, int width, int height) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= width || py >= height)return;
    int pidx = px + width * (py + height * pz);
    px <<= 1; py <<= 1;
    width <<= 1; height <<= 1;
    int p00idx = px + width * (py + height * pz);
    int p01idx = p00idx + 1;
    int p10idx = p00idx + width;
    int p11idx = p10idx + 1;

    int idx = index - 1;
    for (int i = 0; i < texture.channel; i++) {
        float p00 = texture.texture[idx][p00idx * texture.channel + i];
        float p01 = texture.texture[idx][p01idx * texture.channel + i];
        float p10 = texture.texture[idx][p10idx * texture.channel + i];
        float p11 = texture.texture[idx][p11idx * texture.channel + i];

        float p = (p00 + p01 + p10 + p11) * 0.25f;
        texture.texture[index][pidx * texture.channel + i] = p;
    }
}

void Texture::buildMIP(Texture& texture) {
    int i = 1;
    int w = texture.width, h = texture.height;
    void* args[] = { &texture, &i, &w, &h };
    for (; i < texture.miplevel; i++) {
        w >>= 1; h >>= 1;
        dim3 block = getBlock(w, h);
        dim3 grid = getGrid(block, w, h);
        CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(downSampling), grid, block, args, 0, NULL));
    }
}

__global__ void bmpUcharToFloat(unsigned char* data, const Texture texture) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= texture.width || py >= texture.height)return;
    int pidx = px + texture.width * (py + texture.height * pz);

    for (int i = 0; i < texture.channel; i++) {
        texture.texture[0][pidx * texture.channel + i] = (float)data[(pidx + 1) * texture.channel - (i + 1)] / 255.f;
    }
}

void Texture::loadBMP(const char* path, Texture& texture, int miplevel) {
    unsigned char header[54];

    FILE* file = fopen(path, "rb");
    if (!file) {
        ERROR_STRING(Image could not be opened);
        return;
    }
    if (fread(header, 1, 54, file) != 54) {
        ERROR_STRING(Not a correct BMP file);
        return;
    }
    if (header[0] != 'B' || header[1] != 'M') {
        ERROR_STRING(Not a correct BMP file);
        return;
    }
    unsigned int dataPos = *(int*)&(header[0x0A]);
    unsigned int imageSize = *(int*)&(header[0x22]);
    unsigned int width = *(int*)&(header[0x12]);
    unsigned int height = *(int*)&(header[0x16]);
    unsigned int channel = *(int*)&(header[0x1c]) / 8;
    Texture::init(texture, width, height, channel, miplevel);
    if (imageSize == 0)    imageSize = width * height * channel;
    if (dataPos == 0)      dataPos = 54;
    fseek(file, dataPos, SEEK_SET);

    unsigned char* data;
    hipHostMalloc(&data, imageSize * sizeof(unsigned char));
    fread(data, 1, imageSize, file);
    fclose(file);

    unsigned char* dev_data;

    CUDA_ERROR_CHECK(hipMalloc(&dev_data, (size_t)imageSize * sizeof(unsigned char)));
    CUDA_ERROR_CHECK(hipMemcpy(dev_data, data, (size_t)imageSize * sizeof(unsigned char), hipMemcpyHostToDevice));

    dim3 block = getBlock(width, height);
    dim3 grid = getGrid(block, width, height);
    void* args[] = { &dev_data,&texture };
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(bmpUcharToFloat), grid, block, args, 0, NULL));
    CUDA_ERROR_CHECK(hipHostFree(data));
    CUDA_ERROR_CHECK(hipFree(dev_data));
    buildMIP(texture);
}

__global__ void TextureSetColorKernel (const Texture texture, int index, int width, int height, float* color) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= width || py >= height)return;
    int pidx = px + width * (py + height * pz);

    for (int i = 0; i < texture.channel; i++) {
        texture.texture[index][pidx * texture.channel + i] = color[i];
    }
}

void Texture::setColor(Texture& texture, float* color) {
    int w = texture.width, h = texture.height;
    int i = 0;
    float* dev_color;
    CUDA_ERROR_CHECK(hipMalloc(&dev_color, (size_t)texture.channel * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(dev_color, color, (size_t)texture.channel * sizeof(float), hipMemcpyHostToDevice));
    void* args[] = { &texture,&i, &w, &h ,&dev_color };
    for (; i < texture.miplevel; i++) {
        dim3 block = getBlock(w, h);
        dim3 grid = getGrid(block, w, h);
        CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(TextureSetColorKernel), grid, block, args, 0, NULL));
        w >>= 1; h >>= 1;
    }
    CUDA_ERROR_CHECK(hipFree(dev_color));
}

__global__ void TextureLinerKernel (const Texture texture, int index, int width, int height, float w, float b) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= width || py >= height)return;
    int pidx = px + width * (py + height * pz);

    for (int i = 0; i < texture.channel; i++) {
        texture.texture[index][pidx * texture.channel + i] = texture.texture[index][pidx * texture.channel + i] * w + b;
    }
}

void Texture::liner(Texture& texture, float w, float b) {
    int w_ = texture.width, h_ = texture.height;
    int i = 0;
    void* args[] = { &texture,&i, &w_, &h_, &w, &b};
    for (; i < texture.miplevel; i++) {
        dim3 block = getBlock(w_, h_);
        dim3 grid = getGrid(block, w_, h_);
        CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(TextureLinerKernel), grid, block, args, 0, NULL));
        w_ >>= 1; h_ >>= 1;
    }
}

__global__ void normalizeKernel (const Texture texture, int index, int width, int height) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= width || py >= height)return;
    int pidx = px + width * (py + height * pz);

    float s = 0.f;
    for (int i = 0; i < texture.channel; i++) {
        float v = texture.texture[index][pidx * texture.channel + i];
        s += v * v;
    }
    s = 1.f/sqrt(s);
    if (isfinite(s)) {
        for (int i = 0; i < texture.channel; i++) {
            texture.texture[index][pidx * texture.channel + i] *= s;
        }
    }
    else {
        for (int i = 0; i < texture.channel - 1; i++) {
            texture.texture[index][pidx * texture.channel + i] = 0.f;
        }
        texture.texture[index][(pidx + 1) * texture.channel - 1] = 1.f;
    }
}

void Texture::normalize(Texture& texture) {
    int w = texture.width, h = texture.height;
    int i = 0;
    void* args[] = { &texture,&i, &w, &h};
    for (; i < texture.miplevel; i++) {
        dim3 block = getBlock(w, h);
        dim3 grid = getGrid(block, w, h);
        CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(normalizeKernel), grid, block, args, 0, NULL));
        w >>= 1; h >>= 1;
    }
}

__global__ void TextureRandomKernel(const Texture texture, float max, float min, unsigned int seed) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= texture.width || py >= texture.height)return;
    int pidx = px + texture.width * (py + texture.height * pz);

    for (int i = 0; i < texture.channel; i++) {
        texture.texture[0][pidx * texture.channel + i] = min + (max - min) * getUniform(pidx * texture.channel + i,seed,0xf122ba22);
    }
}

void Texture::addRandom(Texture& texture, float max, float min) {
    int w_ = texture.width, h_ = texture.height;
    unsigned int seed = rand();
    dim3 block = getBlock(texture.width, texture.height);
    dim3 grid = getGrid(block, texture.width, texture.height);
    void* args[] = { &texture,&max,&min ,&seed };
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(TextureRandomKernel), grid, block, args, 0, NULL));
    buildMIP(texture);
}

__global__ void TextureClampKernel(const Texture texture, float min, float max) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= texture.width || py >= texture.height)return;
    int pidx = px + texture.width * (py + texture.height * pz);

    for (int i = 0; i < texture.channel; i++) {
        texture.texture[0][pidx * texture.channel + i] = clamp(texture.texture[0][pidx * texture.channel + i], min, max);
    }
}

void Texture::clamp(Texture& texture, float min, float max) {
    int w_ = texture.width, h_ = texture.height;
    dim3 block = getBlock(texture.width, texture.height);
    dim3 grid = getGrid(block, texture.width, texture.height);
    void* args[] = { &texture,&min ,&max};
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(TextureClampKernel), grid, block, args, 0, NULL));
    buildMIP(texture);
}



void TextureGrad::init(TextureGrad& texture, int width, int height, int channel, int miplevel) {
    Texture::init(texture, width, height, channel, miplevel);
    int w = width, h = height;
    for (int i = 0; i < miplevel; i++) {
        CUDA_ERROR_CHECK(hipMalloc(&texture.grad[i], (size_t)w * h * channel * sizeof(float)));
        w >>= 1; h >>= 1;
    }
};

void TextureGrad::clear(TextureGrad& texture) {
    int w = texture.width, h = texture.height;
    for (int i = 0; i < texture.miplevel; i++) {
        CUDA_ERROR_CHECK(hipMemset(texture.grad[i], 0, (size_t)w * h * texture.channel * sizeof(float)));
        w >>= 1; h >>= 1;
    }
}

__global__ void gardAddThrough(const TextureGrad texture, int index, int width, int height) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= width || py >= height)return;
    int pidx = px + width * (py + height * pz);
    px <<= 1; py <<= 1;
    width <<= 1; height <<= 1;
    int p00idx = px + width * (py + height * pz);
    int p01idx = p00idx + 1;
    int p10idx = p00idx + width;
    int p11idx = p10idx + 1;

    int idx = index - 1;
    for (int i = 0; i < texture.channel; i++) {
        float g = texture.grad[index][pidx * texture.channel + i];
        AddNaNcheck(texture.grad[idx][p00idx * texture.channel + i], g);
        AddNaNcheck(texture.grad[idx][p01idx * texture.channel + i], g);
        AddNaNcheck(texture.grad[idx][p10idx * texture.channel + i], g);
        AddNaNcheck(texture.grad[idx][p11idx * texture.channel + i], g);
    }
}

void TextureGrad::gradSumup(TextureGrad& texture) {
    int i = 0;
    int w = texture.width >> (texture.miplevel - 1); int h = texture.height >> (texture.miplevel - 1);
    void* args[] = { &texture, &i, &w, &h };
    for (i = texture.miplevel - 1; i > 0; i--) {
        dim3 block = getBlock(w, h);
        dim3 grid = getGrid(block, w, h);
        CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(gardAddThrough), grid, block, args, 0, NULL));
        w <<= 1; h <<= 1;
    }
}
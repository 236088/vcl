#include "hip/hip_runtime.h"
#include "project.h"

void Project::init(ProjectParams& proj, float* mat, Attribute& vec, bool homogeneous) {
	if (vec.dimention != 3)ERROR_STRING(dimention is not 3);
	proj.kernel.vecNum = vec.vboNum;
	proj.kernel.dimention = homogeneous ? 4 : 3;
	proj.kernel.vec = vec.vbo;
	proj.kernel.mat = mat;
	proj.vao = vec.vao;
	proj.vaoNum = vec.vaoNum;
	CUDA_ERROR_CHECK(hipMalloc(&proj.kernel.out, proj.vecSize()));
}

void Project::init(ProjectParams& proj, float* mat, Attribute& vec, Attribute& out, bool homogeneous) {
	if (vec.dimention != 3)ERROR_STRING(dimention is not 3);
	Attribute::init(out, vec, homogeneous ? 4 : 3);
	proj.kernel.vecNum = vec.vboNum;
	proj.kernel.dimention = out.dimention;
	proj.kernel.vec = vec.vbo;
	proj.kernel.mat = mat;
	proj.vao = vec.vao;
	proj.vaoNum = vec.vaoNum;
	proj.kernel.out = out.vbo;
}

__global__ void ProjectForwardKernel(const ProjectKernelParams proj) {
	int pidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pidx >= proj.vecNum)return;
	float3 v = ((float3*)proj.vec)[pidx];
	for (int i = 0; i < proj.dimention; i++) {
		proj.out[pidx * proj.dimention + i] = proj.mat[i] * v.x + proj.mat[4 + i] * v.y + proj.mat[8 + i] * v.z + proj.mat[12 + i];
	}
}

void Project::forward(ProjectParams& proj) {
	dim3 block = getBlock(proj.kernel.vecNum, 1);
	dim3 grid = getGrid(block, proj.kernel.vecNum, 1);
	void* args[] = { &proj.kernel };
	CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(ProjectForwardKernel), grid, block, args, 0, NULL));
}

void Project::forward(ProjectGradParams& proj) {
	CUDA_ERROR_CHECK(hipMemset(proj.grad.out, 0, proj.vecSize()));
	forward((ProjectParams&)proj);
}

void Project::init(ProjectGradParams& proj, float* mat, AttributeGrad& vec, bool homogeneous) {
	init((ProjectParams&)proj, mat, vec, homogeneous);
	CUDA_ERROR_CHECK(hipMalloc(&proj.grad.out, proj.vecSize()));
	proj.grad.vec = vec.grad;
}

void Project::init(ProjectGradParams& proj, float* mat, float* grad, AttributeGrad& vec, bool homogeneous) {
	init(proj, mat, vec, homogeneous);
	proj.grad.mat = grad;
}

void Project::init(ProjectGradParams& proj, float* mat, float* grad, Attribute& vec, bool homogeneous) {
	init((ProjectParams&)proj, mat, vec, homogeneous);
	CUDA_ERROR_CHECK(hipMalloc(&proj.grad.out, proj.vecSize()));
	proj.grad.mat = grad;
}

void Project::init(ProjectGradParams& proj, float* mat, AttributeGrad& vec, AttributeGrad& out, bool homogeneous) {
	init((ProjectParams&)proj, mat, vec, out, homogeneous);
	proj.grad.out = out.grad;
	proj.grad.vec = vec.grad;
}

__global__ void ProjectBackwardKernel(const ProjectKernelParams proj, const ProjectKernelGradParams grad) {
	int pidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pidx >= proj.vecNum)return;
	for (int i = 0; i < proj.dimention; i++) {
		float g = grad.out[pidx * proj.dimention + i];
		if (grad.vec != nullptr) {
			grad.vec[pidx * 3] += proj.mat[i] * g;
			grad.vec[pidx * 3 + 1] += proj.mat[4 + i] * g;
			grad.vec[pidx * 3 + 2] += proj.mat[8 + i] * g;
		}
		if (grad.mat != nullptr ) {
			grad.mat[i] += proj.vec[pidx * 3] * g;
			grad.mat[4 + i] += proj.vec[pidx * 3 + 1] * g;
			grad.mat[8 + i] += proj.vec[pidx * 3 + 2] * g;
		}
	}
}

void Project::backward(ProjectGradParams& proj) {
	dim3 block = getBlock(proj.kernel.vecNum, 1);
	dim3 grid = getGrid(block, proj.kernel.vecNum, 1);
	void* args[] = { &proj.kernel,&proj.grad};
	CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(ProjectBackwardKernel), grid, block, args, 0, NULL));
}

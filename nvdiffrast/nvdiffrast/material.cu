#include "hip/hip_runtime.h"
#include "material.h"

void NormalAxis::init(NormalAxisParams& norm, RotationParams& rot, RasterizeParams& rast, Attribute& normal) {
    norm.kernel.width = rast.kernel.width;
    norm.kernel.height = rast.kernel.height;
    norm.kernel.depth = rast.kernel.depth;
    norm.kernel.rast = rast.kernel.out;
    norm.kernel.rot = rot.kernel.out;
    norm.kernel.normal = normal.vbo;
    norm.kernel.normalidx = normal.vao;
    CUDA_ERROR_CHECK(hipMalloc(&norm.kernel.out, norm.Size()));
}

void NormalAxis::init(NormalAxisParams& norm, RotationParams& rot, RasterizeParams& rast, Attribute& normal, Attribute& pos, Attribute& texel, TexturemapParams& normalmap) {
    init(norm, rot, rast, normal);
    if (pos.dimention != 3)ERROR_STRING(dimention is not 3);
    norm.kernel.pos = pos.vbo;
    norm.kernel.posidx = pos.vao;
    norm.kernel.texel = texel.vbo;
    norm.kernel.texelidx = texel.vao;
	if (normalmap.kernel.channel != 3)ERROR_STRING(dimention is not 3);
	norm.kernel.normalmap = normalmap.kernel.out;
}

__global__ void NormalAxisForwardKernel(const NormalAxisKernelParams norm) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= norm.width || py >= norm.height || pz >= norm.depth)return;
    int pidx = px + norm.width * (py + norm.height * pz);

    float4 r = ((float4*)norm.rast)[pidx];
    int idx = (int)r.w - 1;
    if (idx < 0) return;

    float3 n2 = ((float3*)norm.normal)[norm.normalidx[idx * 3 + 2]];
    float3 n0 = ((float3*)norm.normal)[norm.normalidx[idx * 3]] - n2;
    float3 n1 = ((float3*)norm.normal)[norm.normalidx[idx * 3 + 1]] - n2;
    float3 N = normalize(n0 * r.x + n1 * r.y + n2);
    if (norm.normalmap == nullptr) {
        float3 normal = make_float3(
            norm.rot[0] * N.x + norm.rot[4] * N.y + norm.rot[8] * N.z,
            -(norm.rot[1] * N.x + norm.rot[5] * N.y + norm.rot[9] * N.z),
            norm.rot[2] * N.x + norm.rot[6] * N.y + norm.rot[10] * N.z
        );
        ((float3*)norm.out)[pidx] = normal;
        return;
    }

    float2 uv2 = ((float2*)norm.texel)[norm.texelidx[idx * 3 + 2]];
    float2 uv0 = ((float2*)norm.texel)[norm.texelidx[idx * 3]] - uv2;
    float2 uv1 = ((float2*)norm.texel)[norm.texelidx[idx * 3 + 1]] - uv2;
    float3 p2 = ((float3*)norm.pos)[norm.posidx[idx * 3 + 2]];
    float3 p0 = ((float3*)norm.pos)[norm.posidx[idx * 3]] - p2;
    float3 p1 = ((float3*)norm.pos)[norm.posidx[idx * 3 + 1]] - p2;

    float3 T = p0 * uv1.y - p1 * uv0.y;
    T = normalize(T - N * dot(N, T));
    float3 B = p1 * uv0.x - p0 * uv1.x;
    B = normalize(B - N * dot(N, B));

    float3 normal = ((float3*)norm.normalmap)[pidx];
    N = normalize(T * normal.x + B * normal.y + N * normal.z);
    ((float3*)norm.out)[pidx] = normalize(make_float3(
        norm.rot[0] * N.x + norm.rot[4] * N.y + norm.rot[8] * N.z,
        -(norm.rot[1] * N.x + norm.rot[5] * N.y + norm.rot[9] * N.z),
        norm.rot[2] * N.x + norm.rot[6] * N.y + norm.rot[10] * N.z
    ));
}

void NormalAxis::forward(NormalAxisParams& norm){
	CUDA_ERROR_CHECK(hipMemset(norm.kernel.out, 0, norm.Size()));
	dim3 block = getBlock(norm.kernel.width, norm.kernel.height);
	dim3 grid = getGrid(block, norm.kernel.width, norm.kernel.height, norm.kernel.depth);
	void* args[] = { &norm.kernel };
	CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(NormalAxisForwardKernel), grid, block, args, 0, NULL));
}



void ViewAxis::init(ViewAxisParams& view, RotationParams& rot, CameraParams& cam, RasterizeParams& rast) {
    view.kernel.width = rast.kernel.width;
    view.kernel.height = rast.kernel.height;
    view.kernel.depth = rast.kernel.depth;
    view.kernel.rot = rot.kernel.out;
    view.kernel.view = cam.kernel.view;
    view.kernel.projection = cam.kernel.projection;
    CUDA_ERROR_CHECK(hipMalloc(&view.kernel.pvinv, 9 * sizeof(float)));
    CUDA_ERROR_CHECK(hipMalloc(&view.kernel.out, view.Size()));
}

__global__ void ViewAxisForwardKernel(const ViewAxisKernelParams view) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= view.width || py >= view.height || pz >= view.depth)return;
    int pidx = px + view.width * (py + view.height * pz);

    float2 screenpos = make_float2(
        (float)px / (float)view.width * 2.f - 1.f,
        (float)py / (float)view.height * 2.f - 1.f
    );

    ((float3*)view.out)[pidx] = normalize(
         ((float3*)view.pvinv)[0] * screenpos.x
        - ((float3*)view.pvinv)[1] * screenpos.y
        + ((float3*)view.pvinv)[2]);
}

void ViewAxis::forward(ViewAxisParams& view) {
    glm::mat3 pvinv = glm::inverse(glm::mat3(*view.kernel.projection) * glm::mat3(*view.kernel.view));
    CUDA_ERROR_CHECK(hipMemcpy(view.kernel.pvinv, &pvinv, 9 * sizeof(float), hipMemcpyHostToDevice));

    dim3 block = getBlock(view.kernel.width, view.kernel.height);
    dim3 grid = getGrid(block, view.kernel.width, view.kernel.height, view.kernel.depth);
    void* args[] = { &view.kernel};
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(ViewAxisForwardKernel), grid, block, args, 0, NULL));
}



void SphericalGaussian::init(SphericalGaussianParams& sg, RasterizeParams& rast, NormalAxisParams& normal, ViewAxisParams& view, TexturemapParams& diffuse, TexturemapParams& roughness, SGBuffer& sgbuf, float ior) {
    sg.kernel.width = rast.kernel.width;
    sg.kernel.height = rast.kernel.height;
    sg.kernel.depth = rast.kernel.depth;
    sg.kernel.channel = sgbuf.channel;

    sg.kernel.rast = rast.kernel.out;

    sg.kernel.normal = normal.kernel.out;
    sg.kernel.view = view.kernel.out;
    sg.kernel.diffuse = diffuse.kernel.out;
    sg.kernel.roughness = roughness.kernel.out;
    sg.kernel.ior = ior;

    sg.kernel.sgnum = sgbuf.num;
    sg.kernel.axis = sgbuf.axis;
    sg.kernel.sharpness = sgbuf.sharpness;
    sg.kernel.amplitude = sgbuf.amplitude;

    CUDA_ERROR_CHECK(hipMalloc(&sg.kernel.out, sg.Size()));
    CUDA_ERROR_CHECK(hipMalloc(&sg.kernel.outDiffenv, sg.Size()));
    CUDA_ERROR_CHECK(hipMalloc(&sg.kernel.outSpecenv, sg.Size()));
}


__global__ void SGForwardKernel(const SphericalGaussianKernelParams sg) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= sg.width || py >= sg.height || pz >= sg.depth)return;
    int pidx = px + sg.width * (py + sg.height * pz);

    float3 v = ((float3*)sg.view)[pidx];

    float4 r = ((float4*)sg.rast)[pidx];
    int idx = (int)r.w - 1;
    if (idx < 0) {
        for (int i = 0; i < sg.sgnum; i++) {
            float3 sgaxis = ((float3*)sg.axis)[i];
            float sgsharpness = sg.sharpness[i];
            float sgvalue = exp(sgsharpness * (dot(v, sgaxis) - 1.f));
            for (int k = 0; k < sg.channel; k++) {
                AddNaNcheck(sg.out[pidx * sg.channel + k], sg.amplitude[i * sg.channel + k] * sgvalue);
            }
        }
        return;
    }
    v = -v;
    float3 n = ((float3*)sg.normal)[pidx];
    float3 p = 2.f * dot(v, n) * n - v;

    float diffamplitude = .32424871f; //=1/(pi*(1-exp(-4))
    float diffsharpness = 2.f;
    float3 diffn = n * diffsharpness;

    float m = sg.roughness[pidx];
    float m2 = m * m;

    float vn = max(dot(v, n),1e-6);
    float vn2 = vn * vn;
    float Gv = 1.f / (vn + sqrt(vn2 + m2 * (1.f - vn2)));

    float pn = max(dot(p, n),1e-6);
    float pn2 = pn * pn;
    float Gp = 1.f / (pn + sqrt(pn2 + m2 * (1.f - pn2)));

    float g = sqrt(sg.ior * sg.ior - 1.f + pn * pn);
    float g_pls_pn = g + pn;
    float g_mns_pn = g - pn;
    float f0 = g_mns_pn / g_pls_pn;
    float f1 = (pn * g_pls_pn - 1.f) / (pn * g_mns_pn + 1.f);
    float F = .5f * f0 * f0 * (1.f + f1 * f1);

    float specamplitude = F*Gp* Gv;
    float specsharpness = .5f / max(m2 * vn, 1e-6);
    float3 specn = p * specsharpness;

    for (int i = 0; i < sg.sgnum; i++) {
        float sgsharpness = sg.sharpness[i];
        float3 sgn = ((float3*)sg.axis)[i] * sgsharpness;
        float diffl = length(diffn + sgn);
        float diffg = (exp(diffl - diffsharpness - sgsharpness) - exp(-diffl - diffsharpness - sgsharpness)) / max(diffl, 1e-6) * diffamplitude * 6.2831853f;
        float specl = length(specn + sgn);
        float specg = (exp(specl - specsharpness - sgsharpness) - exp(-specl - specsharpness - sgsharpness)) / max(specl, 1e-6) * specamplitude * 6.2831853f;
        for (int k = 0; k < sg.channel; k++) {
            float sgamplitude = sg.amplitude[i * sg.channel + k];
            AddNaNcheck(sg.outDiffenv[pidx * sg.channel + k], diffg * sgamplitude);
            AddNaNcheck(sg.outSpecenv[pidx * sg.channel + k], specg * sgamplitude);
        }
    }

    for (int k = 0; k < sg.channel; k++) {
        int idx = pidx * sg.channel + k;
        sg.out[idx] = sg.diffuse[idx] * sg.outDiffenv[idx] + sg.outSpecenv[idx];
    }
}

void SphericalGaussian::forward(SphericalGaussianParams& sg) {
    CUDA_ERROR_CHECK(hipMemset(sg.kernel.out, 0, sg.Size()));
    CUDA_ERROR_CHECK(hipMemset(sg.kernel.outDiffenv, 0, sg.Size()));
    CUDA_ERROR_CHECK(hipMemset(sg.kernel.outSpecenv, 0, sg.Size()));

    dim3 block = getBlock(sg.kernel.width, sg.kernel.height);
    dim3 grid = getGrid(block, sg.kernel.width, sg.kernel.height, sg.kernel.depth);
    void* args[] = { &sg.kernel};
    CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(SGForwardKernel), grid, block, args, 0, NULL));
}

__global__ void SGBackwardKernel(const SphericalGaussianKernelParams sg, const SphericalGaussianGradKernelParams grad) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= sg.width || py >= sg.height || pz >= sg.depth)return;
    int pidx = px + sg.width * (py + sg.height * pz);

    float3 v = ((float3*)sg.view)[pidx];

    float4 r = ((float4*)sg.rast)[pidx];
    int idx = (int)r.w - 1;
    if (idx < 0)  return;

    for (int k = 0; k < sg.channel; k++) {
        int idx = pidx * sg.channel + k;
        grad.diffuse[idx] = grad.out[idx] * sg.outDiffenv[idx];
    }

    float3 n = ((float3*)sg.normal)[pidx];
    float3 p = v - 2.f * dot(v, n) * n;

    float diffamplitude = .32424871f; //=1/(pi*(1-exp(-4))
    float diffsharpness = 2.f;
    float3 diffn = n * diffsharpness;

    float m = sg.roughness[pidx];
    float m2 = m * m;

    float vn = abs(dot(v, n));
    float vn2 = vn * vn;
    float Gv = 1.f / (vn + sqrt(vn2 + m2 * (1.f - vn2)));

    float pn = abs(dot(p, n));
    float pn2 = pn * pn;
    float Gp = 1.f / (pn + sqrt(pn2 + m2 * (1.f - pn2)));

    float g = sqrt(sg.ior * sg.ior - 1.f + pn * pn);
    float g_pls_pn = g + pn;
    float g_mns_pn = g - pn;
    float f0 = g_mns_pn / g_pls_pn;
    float f1 = (pn * g_pls_pn - 1.f) / (pn * g_mns_pn + 1.f);
    float F = .5f * f0 * f0 * (1.f + f1 * f1);

    float specamplitude = F * Gp * Gv;
    float specsharpness = .5f / max(m2 * vn, 1e-6);
    float3 specn = p * specsharpness;

    for (int i = 0; i < sg.sgnum; i++) {
        float sgsharpness = sg.sharpness[i];
        float3 sgn = ((float3*)sg.axis)[i] * sgsharpness;
        float diffl = length(diffn + sgn);
        float diffg = exp(-sgsharpness - diffsharpness) * (exp(diffl) - exp(-diffl)) / max(diffl, 1e-6) * diffamplitude * 6.2831853f;
        float3 specsgn = specn + sgn;
        float specl = length(specsgn);
        float specg = exp(-sgsharpness - specsharpness) * (exp(specl) - exp(-specl)) / max(specl, 1e-6) * specamplitude * 6.2831853f;
        float dgdspecl = exp(-sgsharpness - specsharpness) * ((specl - 1.f) * exp(specl) - (specl + 1.f) * exp(-specl)) / max(dot(specsgn, specsgn), 1e-6) * specamplitude * 6.2831853f;
        float sgamplitudesum = 0.f;
        for (int k = 0; k < sg.channel; k++) {
            sgamplitudesum += sg.amplitude[i * sg.channel + k];
        }

    }
}
#include "hip/hip_runtime.h"
#include "normalcalc.h"

void Normalcalc::init(NormalcalcParams& norm, Attribute& pos, Attribute& normal) {
	if (pos.dimention != 3)ERROR_STRING(dimention is not 3);
	Attribute::init(normal, pos, 3);
	norm.kernel.posNum = pos.vboNum;
	norm.kernel.pos = pos.vbo;
	norm.kernel.vao = pos.vao;
	norm.kernel.vaoNum = pos.vaoNum;
	norm.kernel.out = normal.vbo;
	norm.vaoblock = getBlock(pos.vaoNum, 1);
	norm.vaogrid = getGrid(norm.vaoblock, pos.vaoNum, 1);
	norm.block = getBlock(pos.vboNum, 1);
	norm.grid = getGrid(norm.block, pos.vboNum, 1);
}

__global__ void NormalcalcForwardKernel(const NormalcalcKernelParams norm) {
	int pidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pidx >= norm.vaoNum)return;
	uint3 tri = ((uint3*)norm.vao)[pidx];
	float3 v0 = ((float3*)norm.pos)[tri.x];
	float3 v1 = ((float3*)norm.pos)[tri.y];
	float3 v2 = ((float3*)norm.pos)[tri.z];
	float3 n = normalize(cross(v1 - v0, v2 - v0));
	atomicAdd3(&((float3*)norm.out)[tri.x], n * acos(dot(v1 - v0, v2 - v0)));
	atomicAdd3(&((float3*)norm.out)[tri.y], n * acos(dot(v2 - v1, v0 - v1)));
	atomicAdd3(&((float3*)norm.out)[tri.z], n * acos(dot(v0 - v2, v1 - v2)));
}

__global__ void Normalize(const NormalcalcKernelParams norm) {
	int pidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pidx >= norm.posNum)return;
	((float3*)norm.out)[pidx] = normalize(((float3*)norm.out)[pidx]);
}

void Normalcalc::forward(NormalcalcParams& norm) {
	CUDA_ERROR_CHECK(hipMemset(norm.kernel.out, 0, norm.posSize()));
	void* args[] = { &norm.kernel };
	CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(NormalcalcForwardKernel), norm.vaogrid, norm.vaoblock, args, 0, NULL));
	CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(Normalize), norm.grid, norm.block, args, 0, NULL));
}

void Normalcalc::forward(NormalcalcGradParams& norm) {
	CUDA_ERROR_CHECK(hipMemset(norm.grad.out, 0, norm.posSize()));
	forward((NormalcalcParams&)norm);
}

void Normalcalc::init(NormalcalcGradParams& norm, AttributeGrad& pos, AttributeGrad& normal) {
	init((NormalcalcParams&)norm, pos, normal);
	norm.grad.out = normal.grad;
	norm.grad.pos = pos.grad;
}

__global__ void NormalcalcBackwardKernel(const NormalcalcKernelParams norm, const NormalcalcKernelGradParams grad) {
	int pidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pidx >= norm.vaoNum)return;
}

void Normalcalc::backward(NormalcalcGradParams& norm) {
	void* args[] = { &norm.kernel,&norm.grad };
	CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(NormalcalcBackwardKernel), norm.grid, norm.block, args, 0, NULL));
}
